#include "hip/hip_runtime.h"

#include ""
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// CUDA and CUBLAS functions
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>
//#include <helper_string.h>
#include <stdio.h>
#include "setMatrix.h"
typedef struct {
	int width;
	int height;
	int stride;
	float *elements;
} Matrix_;
#define IDX2C(i,j,ld) (((j)*(ld))+(i));

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
// hipError_t multiMatriWithCuda(float *c, float *a, float *b, int widthA, int heightA, int widthB, int heightB,);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

template<int BLOCK_SIZE>
__global__ void kaisaiMatrixComputation(float *b, float *a){
	int bx = blockIdx.x;
	//int by = blockIdx.y;

	// Thread index 
	int tx = threadIdx.x;
	//int ty = threadIdx.y;

	float Csub = 0;
	// Declaration of the shared memory array as used to store the sum-matrix of A
	__shared__ float As[BLOCK_SIZE];

	As[tx] = a[bx*BLOCK_SIZE + tx];
	Csub = expf(-100 * As[tx]);
	b[bx*BLOCK_SIZE + tx] = Csub;
	return;
}

template<int BLOCK_SIZE>
__global__ void elementWiseDIV(float *c, float *a, float* b){
	int bx = blockIdx.x;
	//int by = blockIdx.y;

	// Thread index 
	int tx = threadIdx.x;
	//int ty = threadIdx.y;

	float Csub = 0;

	// Declaration of the shared memory array as used to store the sum-matrix of A
	__shared__ float As[BLOCK_SIZE];

	// Delcaration of the shared memory array as used to store the sub-matrix of B;
	__shared__ float Bs[BLOCK_SIZE];

	As[tx] = a[bx * BLOCK_SIZE + tx];
	Bs[tx] = b[bx * BLOCK_SIZE + tx];

	if (Bs[tx] > 0 && Bs[tx] < 0.000001){
		Bs[tx] = 0.000001;
	}
	else if (Bs[tx] < 0 && Bs[tx] > -0.000001){
		Bs[tx] = -0.000001;
	}
	
	c[bx * BLOCK_SIZE + tx] = As[tx] / Bs[tx];
	
	return;
}


template<int BLOCK_SIZE>
__global__ void elementWiseMUL(float *c, float *a, float* b){
	int bx = blockIdx.x;
	//int by = blockIdx.y;

	// Thread index 
	int tx = threadIdx.x;
	//int ty = threadIdx.y;

	float Csub = 0;

	// Declaration of the shared memory array as used to store the sum-matrix of A
	__shared__ float As[BLOCK_SIZE];

	// Delcaration of the shared memory array as used to store the sub-matrix of B;
	__shared__ float Bs[BLOCK_SIZE];

	As[tx] = a[bx * BLOCK_SIZE + tx];
	Bs[tx] = b[bx * BLOCK_SIZE + tx];

	
	c[bx * BLOCK_SIZE + tx] = As[tx] * Bs[tx];

	return;
}


template<int BLOCK_SIZE>
__global__ void distancePointToPointCUDA(float *c, float *a, float *b, int hA, int wA, int hB, int wB)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index 
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// index of the first sub-matrix of A processed by the block
	//	int a_begin = BLOCK_SIZE * bx;

	// index of the last sub-matrix of A processed by the block
	//	int a_end = a_begin +  BLOCK_SIZE - 1;

	// Step size used to iterate through the sub-matrices of A
	//	int a_step = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block
	//	int b_begin =  BLOCK_SIZE * by;

	// Index of the last sub-matrix of B proceesed by the block
	//	int b_end = b_begin +  BLOCK_SIZE - 1;

	// Step size used to iterate through the sub-matrices of B
	//	int b_step = BLOCK_SIZE;

	// Csub is used to store the element of the block sub-matrix
	// that is computed by the thread
	float Csub = 0;

	//const int B = wA;

	// Declaration of the shared memory array as used to store the sum-matrix of A
	__shared__ float As[2];

	// Delcaration of the shared memory array as used to store the sub-matrix of B;
	__shared__ float Bs[BLOCK_SIZE * 2];

	// Load the matrices from device memroy 
	// to shared memory; each thread loads 
	// one element of each matrix

#pragma unroll

	for (int i = 0; i < wA; i++){
		As[i] = a[bx * wA + i];

	}

#pragma unroll

	for (int i = 0; i < wA; i++){
		Bs[ty * wA + i] = b[by * BLOCK_SIZE * wB + ty * wB + i];
	}

	// Synchronize to make sure the matrices are loaded

	__syncthreads();

#pragma unroll
	for (int i = 0; i < wA; i++){
		float dif_ = As[i] - Bs[ty * wA + i];
		Csub += dif_ * dif_;
	}

	// Synchronized to make sure that the preceeding 
	// computation is done 

	__syncthreads();
	// Write the block sub- matrix to device memory;
	// eahc thread writes one element

	int c_line = bx;
	int c_col = by * BLOCK_SIZE + ty;
	c[c_line * hB + c_col] = Csub;
}

int distanceCompuation(int block_size, dim3 &dimsA, dim3 &dimsB, float *matrix_A, float *matrix_B, float *matrix_C, float *matrix_D);

int initCuda(){
	int devID = 0;

	hipSetDevice(devID);

	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	if (error != hipSuccess){
		printf("hipGetDevice returned error %s (code %d), line (%d)\n", hipGetErrorString(error), error, __LINE__);
	}

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited){
		fprintf(stderr, "Error: device is runing in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess){
		printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);

	}
	else{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}

	// Use a larger block size for Fermi and above
	int block_size = deviceProp.major < 2 ? 16 : 32;
	return block_size;
}

int main(int argc, char *argv[])
{


	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };
	int DIMENSIONS = 2;

	int num_samplingPoints; // �����������
	int width_originalPoints ; // ԭʼ������ĺ����������
	int height_originalPoints; // ԭʼ������������������
	int num_originalPoints; //
	float *m_samplingPoints; // �������λ�ã�
	float *m_originalPoints; // ԭʼ�ܶȺ�������ɢ�㣻
	float *m_samplingPointsDensity; // ��������ܶȣ�
	float *m_originalPointsDesntiy;

	num_samplingPoints = setNumSamplingPoint();
	width_originalPoints = setWidthOriginalPoint();
	height_originalPoints = setHeightOriginalPoint();
	num_originalPoints = width_originalPoints * height_originalPoints;

	m_samplingPoints = (float *)malloc(num_samplingPoints * DIMENSIONS * sizeof(float));
	m_originalPoints = (float *)malloc(width_originalPoints * height_originalPoints * DIMENSIONS * sizeof(float));
	m_samplingPointsDensity = (float *)malloc(num_samplingPoints * sizeof(float));
	m_originalPointsDesntiy = (float *)malloc(num_originalPoints * sizeof(float));

	setSamplingPoints(m_samplingPoints, num_samplingPoints, DIMENSIONS);
	setOriginalPoints(m_originalPoints, height_originalPoints, width_originalPoints, DIMENSIONS);
	setSamplingPointDensity(m_samplingPointsDensity, num_samplingPoints);
	setSamplingPointDensity(m_originalPointsDesntiy, num_originalPoints);	

	int block_size;

	// ��ʼ��CUDA
	block_size = initCuda();
	
	dim3 dims_sampling_points(num_samplingPoints, 2, 1);
	dim3 dims_original_points(width_originalPoints * height_originalPoints, 2, 1);
    // Add vectors in parallel.
	
	int _result = distanceCompuation(block_size, dims_sampling_points, dims_original_points, m_samplingPoints, m_originalPoints, m_samplingPointsDensity, m_originalPointsDesntiy);

	free(m_samplingPoints);
	free(m_originalPoints);
	free(m_originalPointsDesntiy);
	free(m_samplingPointsDensity);
	exit(_result);

	
}

int distanceCompuation(int block_size, dim3 &dimsA, dim3 &dimsB, float *matrix_A, float *matrix_B, float *matrix_C,float *matrix_D){
	// allocate host memory for original points and sampling points
	
	float *theta = (float *)malloc(sizeof(float));
	*theta = 0.5;
	float *one_minusTheta = (float *)malloc(sizeof(float));
	*one_minusTheta = 1 - *theta;
	float *lamb = (float *)malloc(sizeof(float));
	*lamb = 1;
	float *_R = (float *)malloc(sizeof(float));
	*_R =1;

	float *theta_lambR = (float *)malloc(sizeof(float));
	float *minusOne_divLabR = (float *)malloc(sizeof(float));

	*theta_lambR = *theta * *lamb * *_R;
	*minusOne_divLabR = -1 / *lamb / *_R;


	float stop_U = 0.001; // ���㴫��ƻ������ֹͣ��׼
	float stop_X = 0.001;// ���������ֹͣ��׼

	float alpha = 1.0;
	float beta = 0.0;
	unsigned int _iter = 20;
	float temp_alpha = -1.0;

	unsigned int size_A = dimsA.x * dimsA.y * dimsA.z;
	unsigned int mem_sizeA = sizeof(float) * size_A;
	float *h_A = matrix_A; // �����������

	unsigned int size_B = dimsB.x * dimsB.y * dimsB.z;
	unsigned int mem_sizeB = sizeof(float) * size_B;
	float  *h_B = matrix_B; // ������������꣬ԭʼͼ�������

	unsigned int size_samplingPoint = dimsA.x;
	unsigned int mem_sizeSamplingPoint = sizeof(float) * size_samplingPoint;
	float *h_samplingPointDensity = matrix_C;

	unsigned int size_originalPoint = dimsB.x;
	unsigned int mem_sizeOriginalPoint = sizeof(float) * size_originalPoint;
	float *h_originalPointDensity = matrix_D;

	unsigned int size_transportMatrix = dimsA.x * dimsB.x;
	unsigned int mem_sizeTransportMatrix = size_transportMatrix * sizeof(float);

	float *h_V = (float *)malloc(mem_sizeOriginalPoint);// ���㴫��ƻ�������v��
	for (int i = 0; i < dimsB.x; i++){
		*(h_V + i) = (float)1.0;
	}

	float *h_distanceMatrix;
	float *h_kasaiMatrix; // �������ĸ�˹����
	float *h_transportPlan;// = (float *)malloc(dimsA.x * dimsB.x * sizeof(float));

	// allocate device memory 
	float *d_A, *d_B, *d_distanceMatrix, *d_kasaiMatrix, *d_transportPlan, *d_U, *d_V; // device memory�еı���������d_U, d_V Ϊ�м����
	float *d_kasaiV, *d_kasaiU;
	float *d_samplingPointDensity, *d_originalPointDensity;
	float *d_tempVectorStopCri;
	float *d_diagUKasaiMatrix; /// ��ʱ����
	float *d_transportPlanDensity;///��ʱ����
	float *d_tempSamplPointCoordinate; // ��ʱ����
	hipblasHandle_t handle;
	hipblasStatus_t stat;
	hipError_t error;

	
	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS){
		printf("CUBLAS initialization failed\n");
		exit(EXIT_FAILURE);
	}
	
	error = hipMalloc((void**)&d_A, mem_sizeA);

	if (error != hipSuccess){
		printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString, error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_B, mem_sizeB);

	if (error != hipSuccess){
		printf("hipMalloc d_B returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_distanceMatrix, mem_sizeTransportMatrix);
	if (error != hipSuccess){
		printf("hipMalloc d_distanceMatrix returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_kasaiMatrix, mem_sizeTransportMatrix);
	if (error != hipSuccess){
		printf("hipMalloc d_kasaiMatrix returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_U, mem_sizeSamplingPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_U returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_V, mem_sizeOriginalPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_V returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}
	
	error = hipMalloc((void**)&d_kasaiV, mem_sizeSamplingPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_kasaiV returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_kasaiU, mem_sizeOriginalPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_kasaiU returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_samplingPointDensity, mem_sizeSamplingPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_samplingPointDensity returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_originalPointDensity, mem_sizeOriginalPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_originalPointDensity returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_diagUKasaiMatrix, mem_sizeTransportMatrix);
	if (error != hipSuccess){
		printf("hipMalloc d_diagUKasaiMatrix returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_transportPlan, mem_sizeTransportMatrix);
	if (error != hipSuccess){
		printf("hipMalloc d_transportPlan returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_transportPlanDensity, mem_sizeTransportMatrix);
	if (error != hipSuccess){
		printf("hipMalloc d_transportPlanDensity returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_tempSamplPointCoordinate, mem_sizeA);
	if (error != hipSuccess){
		printf("hipMalloc d_tempSamplPointCoordinate returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_tempVectorStopCri, mem_sizeOriginalPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_tempVectorStopCri returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// copy host memory to device
	error = hipMemcpy(d_A, h_A, mem_sizeA, hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("hipMemcpy (d_A, h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_B, h_B, mem_sizeB, hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("hipMemcpy (d_B, h_B) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_V, h_V, mem_sizeOriginalPoint, hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("hipMemcpy (d_V, h_V) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_samplingPointDensity, h_samplingPointDensity, mem_sizeSamplingPoint, hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("hipMemcpy (d_samplingPointDensity, h_samplingPointDensity) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_originalPointDensity, h_originalPointDensity, mem_sizeOriginalPoint, hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("hipMemcpy (d_originalPointDensity, h_originalPointDensity) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// ����event
	hipEvent_t start;
	error = hipEventCreate(&start);

	if (error != hipSuccess){
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess){
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess){
		fprintf(stderr, " Failed to record start evern (error code %s)! \n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float *stop_valueU = (float *)malloc(sizeof(float));
	float *stop_valueX = (float *)malloc(sizeof(float));

	*stop_valueU = 999999;
	*stop_valueX = 999999;

	//block_size = 32;
	dim3 threads(1, block_size, 1);
	dim3 grid(dimsA.x, dimsB.x / block_size, 1);

	// ����������
	if (block_size == 16){
		distancePointToPointCUDA<16> << <grid, threads >> >(d_distanceMatrix, d_A, d_B, dimsA.x, dimsA.y, dimsB.x, dimsB.y);
	}
	else{
		distancePointToPointCUDA<32> << <grid, threads >> >(d_distanceMatrix, d_A, d_B, dimsA.x, dimsA.y, dimsB.x, dimsB.y);
	}
	// ͬ������
	hipDeviceSynchronize();
	/// ����distancePointToPointCUDA�Ƿ���ȷ
/*	h_distanceMatrix = (float *)malloc(mem_sizeTransportMatrix);
	error = hipMemcpy(h_distanceMatrix, d_distanceMatrix, mem_sizeTransportMatrix, hipMemcpyDeviceToHost);
	if (error != hipSuccess){
		printf("hipMemcpy (h_distanceMatrix, d_distanceMatrix) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}
	// cpu ����������
	float *c_C = (float *)malloc(mem_sizeTransportMatrix);
	for (int i = 0; i < dimsA.x; i++){
		for (int j = 0; j < dimsB.x; j++){
			float diff_x;
			float diff_y;
			diff_x = *(h_A + i * dimsA.y) - *(h_B + j*dimsB.y);
			diff_y = *(h_A + i * dimsA.y + 1) - *(h_B + j*dimsB.y + 1);
			*(c_C + dimsB.x * i + j) = diff_x * diff_x + diff_y * diff_y;
		}
	}
	printf("The distance matrix: GPU  CPU.\n");
	for (int i = 0; i < dimsA.x * dimsB.x; i++){
		printf("The distance matrix: %f  %f \n", h_distanceMatrix[i], c_C[i]);
	}

	float diff_ = 0;
	for (int i = 0; i < dimsA.x; i++){
		for (int j = 0; j < dimsB.x; j++)
			diff_ += abs(h_distanceMatrix[i * dimsB.x + j] - c_C[i*dimsB.x + j]);

	}
	printf("The difference distance computation between results of CPU and GPU is %f.\n", diff_);
	//	free(c_C);
*/	

	float *h_kasaiV = (float *)malloc(mem_sizeSamplingPoint);
	float *h_kasaiU = (float *)malloc(mem_sizeOriginalPoint);
	//float *h_V = (float *)malloc(mem_sizeOriginalPoint);
	float *h_U = (float *)malloc(mem_sizeSamplingPoint);

	while (*stop_valueX > stop_X){		

		// ����Kasai����
		threads.x = block_size;
		threads.y = 1;
		threads.z = 1;
		grid.x = size_transportMatrix / threads.x;
		grid.y = 1;
		grid.z = 1;

		if (block_size == 16){
			kaisaiMatrixComputation<4> << <grid, threads >> >(d_kasaiMatrix, d_distanceMatrix);
		}
		else{
			kaisaiMatrixComputation<32> << <grid, threads >> >(d_kasaiMatrix, d_distanceMatrix);
		}
		// ͬ������
		hipDeviceSynchronize();
		/// �����Ƿ���ȷ
/*		h_kasaiMatrix = (float *)malloc(mem_sizeTransportMatrix);
		error = hipMemcpy(h_kasaiMatrix, d_kasaiMatrix, mem_sizeTransportMatrix, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
			printf("hipMemcpy (h_distanceMatrix, d_distanceMatrix) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
			exit(EXIT_FAILURE);
		}
		float *check_kasaiMatrix = (float *)malloc(mem_sizeTransportMatrix);
		printf("kasaiMatrix on GPU and CPU \n ");
		for (int i = 0; i < size_transportMatrix; i++){
			check_kasaiMatrix[i] = exp(-c_C[i]);
			printf("%f  %f \n", h_kasaiMatrix[i], check_kasaiMatrix[i]);

		}
		diff_ = 0;
		for (int i = 0; i < size_transportMatrix; i++){
			diff_ += abs(check_kasaiMatrix[i] - h_kasaiMatrix[i]);
		}
		printf("The difference between kasai Matrix of CPU and GPU is %f.\n", diff_);
		//	free(check_kasaiMatrix);
*/
		// ���㴫��ƻ�����
						
		float diff_stopValueU = *stop_valueU;
		float stop_valueZero = *stop_valueU;
		while (diff_stopValueU > stop_U){
			for (int i = 0; i < _iter; i++){
				//// d_kasaiMatrix ��һ��size_original x size_sampling �ľ��� d_V ��һ��size_original������
				//  d_kasaiV ��һ�� size_sampling ������
				stat = hipblasSgemv(handle, HIPBLAS_OP_T, size_originalPoint, size_samplingPoint, &alpha, d_kasaiMatrix, size_originalPoint, d_V, 1, &beta, d_kasaiV, 1);
				if (stat != HIPBLAS_STATUS_SUCCESS){
					printf("hipblasSgemv failed\n");
					exit(EXIT_FAILURE);
				}
				// ͬ������
				hipDeviceSynchronize();				
				
				// �����ȷ��
/*				float *check_kasaiV = (float *)malloc(mem_sizeSamplingPoint);
				printf("kasaiV vector: GPU  CPU\n");
				for (int i = 0; i < size_samplingPoint; i++){
					float temp_ = 0;
					for (int j = 0; j < size_originalPoint; j++){
						temp_ += check_kasaiMatrix[i*size_originalPoint + j] * h_V[j];
					}
					check_kasaiV[i] = temp_;
					printf("KasaiV vector: %f  %f\n", h_kasaiV[i], check_kasaiV[i]);
				}
				diff_ = 0;
				for (int i = 0; i < size_samplingPoint; i++){
					diff_ += abs(h_kasaiV[i] - check_kasaiV[i]);
				}
				printf("The differenc of kasaiV vector: %f\n", diff_);
				//	free(check_kasaiV);
*/
				threads.x = block_size;
				threads.y = 1;
				threads.z = 1;
				grid.x = size_samplingPoint / threads.x;
				grid.y = 1;
				grid.z = 1;
				elementWiseDIV<32> << <grid, threads >> >(d_U, d_samplingPointDensity, d_kasaiV);
				// ͬ������
				hipDeviceSynchronize();

			/*	error = hipMemcpy(h_U, d_U, mem_sizeSamplingPoint, hipMemcpyDeviceToHost);

				for (int i = 0; i < size_samplingPoint; i++){

					printf("h_U vector: %f \n", h_U[i]);
				}
				*/

				stat = hipblasSgemv(handle, HIPBLAS_OP_N, size_originalPoint, size_samplingPoint, &alpha, d_kasaiMatrix, size_originalPoint, d_U, 1, &beta, d_kasaiU, 1);
				if (stat != HIPBLAS_STATUS_SUCCESS){
					printf("hipblasSdot failed\n");
					exit(EXIT_FAILURE);
				}
				// ͬ������
				hipDeviceSynchronize();

			/*	error = hipMemcpy(h_kasaiU, d_kasaiU, mem_sizeOriginalPoint, hipMemcpyDeviceToHost);
				if (error != hipSuccess){
					printf("hipMemcpy (h_kasaiV, d_kasaiV) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
					exit(EXIT_FAILURE);
				}  */

		//		for (int i = 0; i < size_originalPoint; i++){

		//			printf("KasaiU vector: %f \n", h_kasaiU[i]);
		//		}
				
				threads.x = block_size;
				threads.y = 1;
				threads.z = 1;
				grid.x = size_originalPoint / threads.x;
				grid.y = 1;
				grid.z = 1;
				elementWiseDIV<32> << <grid, threads >> >(d_V, d_originalPointDensity, d_kasaiU);
				// ͬ������
				hipDeviceSynchronize();
			}

			stat = hipblasSgemv(handle, HIPBLAS_OP_N, size_originalPoint, size_samplingPoint, &alpha, d_kasaiMatrix, size_originalPoint, d_U, 1, &beta, d_kasaiU, 1);
			if (stat != HIPBLAS_STATUS_SUCCESS){
				printf("hipblasSdot failed\n");
				exit(EXIT_FAILURE);
			}
			// ͬ������
			hipDeviceSynchronize();			

			threads.x = block_size;
			threads.y = 1;
			threads.z = 1;
			grid.x = size_originalPoint / threads.x;
			grid.y = 1;
			grid.z = 1;
			elementWiseMUL<32> << <grid, threads >> >(d_tempVectorStopCri, d_V, d_kasaiU);
			hipDeviceSynchronize();

			stat = hipblasSaxpy(handle, size_originalPoint, &temp_alpha, d_originalPointDensity, 1, d_tempVectorStopCri, 1);
			if (stat != HIPBLAS_STATUS_SUCCESS){
				printf("hipblasSdot failed\n");
				exit(EXIT_FAILURE);
			}
			// ͬ������
			hipDeviceSynchronize();

			/// ����u v ��ֵֹͣ
			stat = hipblasSnrm2(handle, size_originalPoint, d_tempVectorStopCri, 1, stop_valueU);
			diff_stopValueU = abs(stop_valueZero - *stop_valueU);
			stop_valueZero = *stop_valueU;
			printf("valueU: %f  %f \n",*stop_valueU, diff_stopValueU);
			// ͬ������
			hipDeviceSynchronize();

		}

/*
		float *h_U = (float *)malloc(mem_sizeSamplingPoint);
		error = hipMemcpy(h_U, d_U, mem_sizeSamplingPoint, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
			printf("hipMemcpy (h_U, d_U) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
			exit(EXIT_FAILURE);
		}

		float *h_kasaiU = (float *)malloc(mem_sizeOriginalPoint);
		error = hipMemcpy(h_kasaiU, d_kasaiU, mem_sizeOriginalPoint, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
			printf("hipMemcpy (h_kasaiU, d_kasaiU) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
			exit(EXIT_FAILURE);
		}

		error = hipMemcpy(h_V, d_V, mem_sizeOriginalPoint, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
			printf("hipMemcpy (h_V, d_V) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
			exit(EXIT_FAILURE);
		}




		float *check_U = (float *)malloc(mem_sizeSamplingPoint);
		printf("temp vector U: GPU    CPU\n");
		for (int i = 0; i < size_samplingPoint; i++){
			check_U[i] = h_samplingPointDensity[i] / h_kasaiV[i];
			printf("%f  %f\n", h_U[i], check_U[i]);
		}

		diff_ = 0;
		for (int i = 0; i < size_samplingPoint; i++){
			diff_ += abs(h_U[i] - check_U[i]);
		}
		printf("the difference of vector U: %f\n", diff_);

		free(check_U);


		float *check_kasaiU = (float *)malloc(mem_sizeOriginalPoint);
		printf("kasaiU vector: GPU  CPU\n");
		for (int i = 0; i < size_originalPoint; i++){
			float temp_ = 0;
			for (int j = 0; j < size_samplingPoint; j++){
				temp_ += h_kasaiMatrix[j*size_originalPoint + i] * h_U[j];
			}
			check_kasaiU[i] = temp_;
			printf("KasaiU vector: %f  %f\n", h_kasaiU[i], check_kasaiU[i]);
		}
		diff_ = 0;
		for (int i = 0; i < size_originalPoint; i++){
			diff_ += abs(h_kasaiU[i] - check_kasaiU[i]);
		}
		printf("The differenc of kasaiU vector: %f\n", diff_);
		free(check_kasaiU);


		float *check_V = (float *)malloc(mem_sizeOriginalPoint);
		printf("temp vector V: GPU    CPU\n");
		for (int i = 0; i < size_originalPoint; i++){
			check_V[i] = h_originalPointDensity[i] / h_kasaiU[i];
			printf("%f  %f\n", h_V[i], check_V[i]);
		}

		diff_ = 0;
		for (int i = 0; i < size_originalPoint; i++){
			diff_ += abs(h_V[i] - check_V[i]);
		}
		printf("the difference of vector V: %f\n", diff_);

		free(check_V);
*/
		///���㴫��ƻ�����
		// ����hipblasSdgmm�����Ծ���û��op���������������൱��ת�õļ��㣬 A= BCD   AT = DT CT BT (T��ʾת��)�ر�С��
		// �Դ��м������ d_transportMatrix ���󣬸պ���һ��size_originalPoint x size_samplingPoint  �Ҹպ��ǰ��������õľ���
		hipblasSdgmm(handle, HIPBLAS_SIDE_LEFT, size_originalPoint, size_samplingPoint, d_kasaiMatrix, size_originalPoint, d_V, 1, d_diagUKasaiMatrix, size_originalPoint);
		// ͬ������
		hipDeviceSynchronize();
		
		hipblasSdgmm(handle, HIPBLAS_SIDE_RIGHT, size_originalPoint, size_samplingPoint, d_diagUKasaiMatrix, size_originalPoint, d_U, 1, d_transportPlan, size_originalPoint);
		// ͬ������
		hipDeviceSynchronize();
		// �˶���ȷ��

/*		h_transportPlan = (float *)malloc(mem_sizeTransportMatrix);

		error = hipMemcpy(h_transportPlan, d_transportPlan, mem_sizeTransportMatrix, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
			printf("hipMemcpy (h_transportPlan, d_transportPlan) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
			exit(EXIT_FAILURE);
		}
*/
/* 
		float *check_transportPlan = (float *)malloc(mem_sizeTransportMatrix);
		float *ch_transportPlan = (float *)malloc(mem_sizeTransportMatrix);

		for (int i = 0; i < size_samplingPoint; i++){
			for (int j = 0; j < size_originalPoint; j++){
				ch_transportPlan[i*size_originalPoint + j] = h_U[i] * check_kasaiMatrix[i*size_originalPoint + j];
			}
		}
		for (int i = 0; i < size_samplingPoint; i++){
			for (int j = 0; j < size_originalPoint; j++){
				check_transportPlan[i*size_originalPoint + j] = ch_transportPlan[i*size_originalPoint + j] * h_V[j];
			}
		}

		printf("Transport plan matrix: GPU  CPU \n");
		for (int i = 0; i < size_transportMatrix; i++){
			printf("Transport plan matrix: %f  %f\n", h_transportPlan[i], check_transportPlan[i]);
		}

	*/
		/*for (int i = 0; i < size_samplingPoint; i++){
			for (int j = 0; j < size_originalPoint; j++){
			printf("  %f  ", h_transportPlan[i*size_originalPoint + j]);
			}
			printf("\n");
			} */


//		printf("Transport plan matrix: CPU\n");
		/*	for (int i = 0; i < size_samplingPoint; i++){
				for (int j = 0; j < size_originalPoint; j++){
				printf("  %f  ", check_transportPlan[i*size_originalPoint + j]);
				}
				printf("\n");
				}*/

		//free(check_transportPlan);
		//free(ch_transportPlan);

		/// ��������ֵ����,����Ϊ��������һ����Y��ԽǾ���Ļ��֣��ڶ����Ǽ���ԭʼ��������µľ����������������Ǽ��������ͣ����ֳ�������С��
		hipblasSdgmm(handle, HIPBLAS_SIDE_RIGHT, size_originalPoint, size_samplingPoint, d_transportPlan, size_originalPoint, d_samplingPointDensity, 1, d_transportPlanDensity, size_originalPoint);
		// ͬ������
		hipDeviceSynchronize();

		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dimsA.y, size_samplingPoint, size_originalPoint, &alpha, d_B, dimsA.y, d_transportPlanDensity, size_originalPoint, &beta, d_tempSamplPointCoordinate, dimsA.y);
		// ͬ������
		hipDeviceSynchronize();
/*
		float *h_transportPlanDensity = (float *)malloc(mem_sizeTransportMatrix);
		error = hipMemcpy(h_transportPlanDensity, d_transportPlanDensity, mem_sizeTransportMatrix, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
			printf("hipMemcpy (h_transportPlanDensity, d_transportPlanDensity) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
			exit(EXIT_FAILURE);
		}

		float *h_tempSamplPointCoordinate = (float *)malloc(mem_sizeA);
		error = hipMemcpy(h_tempSamplPointCoordinate, d_tempSamplPointCoordinate, mem_sizeA, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
			printf("hipMemcpy (h_tempSamplPointCoordinate, d_tempSamplPointCoordinate) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
			exit(EXIT_FAILURE);
		}
*/
/*
		// �˶���ȷ�� h_transportPlanDensity
		float *h_transportPlanDensityT = (float *)malloc(mem_sizeTransportMatrix);
		for (int i = 0; i < size_samplingPoint; i++){
			for (int j = 0; j < size_originalPoint; j++){
				h_transportPlanDensityT[j*size_samplingPoint + i] = h_transportPlanDensity[i*size_originalPoint + j];
			}
		}


		float *check_transportPlanDensity = (float *)malloc(mem_sizeTransportMatrix);
		float *check_transportPlanDensityT = (float *)malloc(mem_sizeTransportMatrix);
		float *check_tempSamplPointCoordinate = (float *)malloc(mem_sizeA);
		float *check_transportPlanT = (float *)malloc(mem_sizeTransportMatrix);
		for (int i = 0; i < size_samplingPoint; i++){
			for (int j = 0; j < size_originalPoint; j++){
				check_transportPlanT[j*size_samplingPoint + i] = check_transportPlan[i*size_originalPoint + j];
			}
		}

		for (int i = 0; i < size_originalPoint; i++){
			for (int j = 0; j < size_samplingPoint; j++){
				check_transportPlanDensity[i*size_samplingPoint + j] = check_transportPlanT[i*size_samplingPoint + j] * h_samplingPointDensity[j];
			}
		}

		for (int i = 0; i < size_originalPoint; i++){
			for (int j = 0; j < size_samplingPoint; j++){
				check_transportPlanDensityT[j* size_originalPoint + i] = check_transportPlanDensity[i*size_samplingPoint + j];
			}
		}

		printf("Tempt matrix h_transportPlanDensity: GPU   CPU\n");
		for (int i = 0; i < size_transportMatrix; i++){
			printf("Tempt matrix h_transportPlanDensity:  %f  %f\n", h_transportPlanDensity[i], check_transportPlanDensityT[i]);
		}

*/
		/*	for (int i = 0; i < size_samplingPoint; i++){
		for (int j = 0; j < size_originalPoint; j++){
		printf("  %f   ", h_transportPlanDensity[i*size_originalPoint + j]);
		}
		printf("\n");
		}

		for (int i = 0; i < size_samplingPoint; i++){
		for (int j = 0; j < size_originalPoint; j++){
		printf("  %f   ", check_transportPlanDensity[i*size_originalPoint + j]);
		}
		printf("\n");
		}
		*/
		// �˶���ȷ�� ����
/*
		float *h_BT = (float *)malloc(mem_sizeB);
		float *check_ordinate = (float*)malloc(mem_sizeA);
		for (int i = 0; i < size_originalPoint; i++){
			for (int j = 0; j < dimsB.y; j++){
				h_BT[j*size_originalPoint + i] = h_B[i*dimsB.y + j];
			}
		}


		for (int i = 0; i < dimsB.y; i++){
			for (int j = 0; j < size_samplingPoint; j++){
				check_ordinate[i * size_samplingPoint + j] = 0;
				for (int k = 0; k < size_originalPoint; k++){
					check_ordinate[i * size_samplingPoint + j] += h_BT[i * size_originalPoint + k] * check_transportPlanDensity[k * size_samplingPoint + j];
				}
			}
		}



		float *h_tempSamplPointCoordinateT = (float *)malloc(mem_sizeA);
		for (int i = 0; i < dimsA.y; i++){
			for (int j = 0; j < size_samplingPoint; j++){
				h_tempSamplPointCoordinateT[i * size_samplingPoint + j] = h_tempSamplPointCoordinate[j * dimsA.y + i];
			}
		}

		printf("Cordinate Y x Kasai x diag��gi�� on GPU  CPU\n");
		for (int i = 0; i < dimsA.y * size_samplingPoint; i++){
			printf("Coordinate: %f  %f \n", h_tempSamplPointCoordinateT[i], check_ordinate[i]);
		}

*/
		hipblasSaxpy(handle, size_samplingPoint * dimsA.y, minusOne_divLabR, d_A, 1, d_tempSamplPointCoordinate, 1);
		// ͬ������
		hipDeviceSynchronize();

		hipblasSaxpy(handle, size_samplingPoint * dimsA.y, theta_lambR, d_tempSamplPointCoordinate, 1, d_A, 1);
		// ͬ������
		hipDeviceSynchronize();

/*		float *h_samplPointCoordinate = (float *)malloc(mem_sizeA);
		error = hipMemcpy(h_samplPointCoordinate, d_A, mem_sizeA, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
			printf("hipMemcpy (h_tempSamplPointCoordinate, d_tempSamplPointCoordinate) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
			exit(EXIT_FAILURE);
		}

*/
/*
		float *h_AT = (float *)malloc(mem_sizeA);
		for (int i = 0; i < size_samplingPoint; i++){
			for (int j = 0; j < dimsA.y; j++){
				h_AT[j*size_samplingPoint + i] = h_A[i*dimsA.y + j];
			}
		}

		for (int i = 0; i < size_samplingPoint * dimsA.y; i++){
			float temp_ = check_ordinate[i];
			check_ordinate[i] = *one_minusTheta * h_AT[i] + *theta_lambR * temp_;
		}

		float *h_samplPointCoordinateT = (float *)malloc(mem_sizeA);
		for (int i = 0; i < size_samplingPoint; i++){
			for (int j = 0; j < dimsA.y; j++){
				h_samplPointCoordinateT[j * size_samplingPoint + i] = h_samplPointCoordinate[i * dimsA.y + j];
			}
		}

		printf("the updated coordinate: GPU  CPU \n");
		for (int i = 0; i < size_samplingPoint*dimsA.y; i++){
			printf("the updated coordinate: %f  %f \n", h_samplPointCoordinateT[i], check_ordinate[i]);
		}
*/
		// ���¾������
		//block_size = 4;
	//	dim3 threads(1, block_size, 1);
  //		dim3 grid(dimsA.x, dimsB.x / block_size, 1);

		threads.x = 1;
		threads.y = block_size;
		threads.z = 1;
		grid.x = dimsA.x;
		grid.y = dimsB.x / block_size;
		grid.z = 1;


		// ����������
		if (block_size == 16){
			distancePointToPointCUDA<4> << <grid, threads >> >(d_distanceMatrix, d_A, d_B, dimsA.x, dimsA.y, dimsB.x, dimsB.y);
		}
		else{
			distancePointToPointCUDA<32> << <grid, threads >> >(d_distanceMatrix, d_A, d_B, dimsA.x, dimsA.y, dimsB.x, dimsB.y);
		}
		// ͬ������
		hipDeviceSynchronize();

		//// ���㴫����۵�
		hipblasSdot(handle, size_transportMatrix, d_distanceMatrix, 1, d_transportPlan, 1, stop_valueX);
		// ͬ������
		printf("������ۣ� %f\n", *stop_valueX);
		hipDeviceSynchronize();

	}

	/// ������ĵ�
	float *h_samplPointCoordinate = (float *)malloc(mem_sizeA);
	error = hipMemcpy(h_samplPointCoordinate, d_A, mem_sizeA, hipMemcpyDeviceToHost);
	if (error != hipSuccess){
		printf("hipMemcpy (h_tempSamplPointCoordinate, d_tempSamplPointCoordinate) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}
	
	/// 

	// CUBLAS handle
	
	stat = hipblasDestroy(handle);
	if (stat != HIPBLAS_STATUS_SUCCESS){
		printf("hipblasDestroy failed\n");
		exit(EXIT_FAILURE);
	}	

	// Record the stop event
	error = hipEventRecord(stop, NULL);
	if (error != hipSuccess){
		fprintf(stderr, " Failed to record stop event ( error code %s)! \n", hipGetErrorString(error));
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);

	if (error != hipSuccess){
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)! \n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	
	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);

	if (error != hipSuccess){
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	
/*	for (int i = 0; i < dimsA.x; i++){
		for (int j = 0; j < dimsA.y; j++){	
			printf("%f  ", h_A[j*dimsA.x + i]);
		}
		printf("\n");
	}
*/
/*	for (int i = 0; i < dimsA.x * dimsA.y; i++){
		
			printf("%f  ", h_A[ i]);
		}
		printf("\n");

	
	for (int i = 0; i < dimsB.x; i++){
		for (int j = 0; j < dimsB.y; j++){		
			printf("%f  ", h_B[ i * dimsB.y + j]);
		}
		printf("\n");
	}

	for (int i = 0; i < dimsA.x; i++){
		for (int j = 0; j < dimsB.x; j++){
	//		printf("%f  ", h_C[i*dimsB.x + j]);
		}
		printf("\n");
	}

	for (int i = 0; i < dimsB.x; i++){
		for (int j = 0; j < dimsB.y; j++){
//			printf("%f  ", h_D[i * dimsB.y + j]);
		}
		printf("\n");
	}
*/
	// Check  the result
	/*
	float *c_C = (float *)malloc(mem_sizeC);
	for (int i = 0; i < dimsA.x; i++){
		for (int j = 0; j < dimsB.x; j++){
			float diff_x;
			float diff_y;
			diff_x = *(h_A + i * dimsA.y) - *(h_B + j*dimsB.y );
			diff_y = *(h_A + i * dimsA.y + 1) - *(h_B + j*dimsB.y + 1);
			*(c_C + dimsB.x * i + j) = diff_x * diff_x + diff_y * diff_y;
		}
	}

	float diff_ = 0;
	for (int i = 0; i < dimsA.x; i++){
		for (int j = 0; j < dimsB.x; j++)
			diff_ += ( h_C[i * dimsB.x + j] - c_C[i*dimsB.x + j]) * (h_C[i * dimsB.x + j] - c_C[i*dimsB.x + j] );

	}
	*/

	//printf("The difference between results of CPU and GPU is %f.\n", diff_);

	//printf("The dot product of h_B is %f\n", result_);

	//Clean up memory
	//free(h_A);
	//free(h_B);
	//free(h_C);
	//free(h_D);
	free(h_transportPlan);
	hipFree(d_A);
	hipFree(d_B);
	//hipFree(d_C);

	printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits

	hipDeviceReset();

	return EXIT_SUCCESS;

}


// set variables for distance compuation between points
void setVariableForDistanceComputation(){




	return;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
	int *dev_d = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_d, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);



	addKernel << <1, size >> >(dev_d, dev_c, dev_b);


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_d, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
	
	hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

/*
hipError_t multiMatriWithCuda(float *c, float *a, float *b, int widthA, int heightA, int widthB, int heightB)
{
	float2 *dev_a = 0;
	float2 *dev_b = 0;
	float2 *dev_c = 0;
	
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, heightA * heightB * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, heightA * widthA * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, heightB * widthB * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}



	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, widthA * heightA * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, widthB * heightB * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_d, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
*/