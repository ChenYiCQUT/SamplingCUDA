#include "hip/hip_runtime.h"

#include ""
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// CUDA and CUBLAS functions
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>
//#include <helper_string.h>
#include <stdio.h>
#include "setMatrix.h"
typedef struct {
	int width;
	int height;
	int stride;
	float *elements;
} Matrix_;
#define IDX2C(i,j,ld) (((j)*(ld))+(i));

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
// hipError_t multiMatriWithCuda(float *c, float *a, float *b, int widthA, int heightA, int widthB, int heightB,);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

template<int BLOCK_SIZE>
__global__ void kaisaiMatrixComputation(float *b, float *a){
	int bx = blockIdx.x;
	//int by = blockIdx.y;

	// Thread index 
	int tx = threadIdx.x;
	//int ty = threadIdx.y;

	float Csub = 0;
	// Declaration of the shared memory array as used to store the sum-matrix of A
	__shared__ float As[BLOCK_SIZE];

	As[tx] = a[bx*BLOCK_SIZE + tx];
	Csub = expf(-1*As[tx]);
	b[bx*BLOCK_SIZE + tx] = Csub;
	return;
}

template<int BLOCK_SIZE>
__global__ void elementWiseDIV(float *c, float *a, float* b){
	int bx = blockIdx.x;
	//int by = blockIdx.y;

	// Thread index 
	int tx = threadIdx.x;
	//int ty = threadIdx.y;

	float Csub = 0;

	// Declaration of the shared memory array as used to store the sum-matrix of A
	__shared__ float As[BLOCK_SIZE];

	// Delcaration of the shared memory array as used to store the sub-matrix of B;
	__shared__ float Bs[BLOCK_SIZE];

	As[tx] = a[bx * BLOCK_SIZE + tx];
	Bs[tx] = b[bx * BLOCK_SIZE + tx];

	if (Bs[tx] > 0 && Bs[tx] < 0.000001){
		Bs[tx] = 0.000001;
	}
	else if (Bs[tx] < 0 && Bs[tx] > -0.000001){
		Bs[tx] = -0.000001;
	}
	c[bx * BLOCK_SIZE + tx] = As[tx] / Bs[tx];

	return;

}

template<int BLOCK_SIZE>
__global__ void distancePointToPointCUDA(float *c, float *a, float *b, int hA, int wA, int hB, int wB)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index 
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// index of the first sub-matrix of A processed by the block
	//	int a_begin = BLOCK_SIZE * bx;

	// index of the last sub-matrix of A processed by the block
	//	int a_end = a_begin +  BLOCK_SIZE - 1;

	// Step size used to iterate through the sub-matrices of A
	//	int a_step = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block
	//	int b_begin =  BLOCK_SIZE * by;

	// Index of the last sub-matrix of B proceesed by the block
	//	int b_end = b_begin +  BLOCK_SIZE - 1;

	// Step size used to iterate through the sub-matrices of B
	//	int b_step = BLOCK_SIZE;

	// Csub is used to store the element of the block sub-matrix
	// that is computed by the thread
	float Csub = 0;

	//const int B = wA;

	// Declaration of the shared memory array as used to store the sum-matrix of A
	__shared__ float As[2];

	// Delcaration of the shared memory array as used to store the sub-matrix of B;
	__shared__ float Bs[BLOCK_SIZE * 2];

	// Load the matrices from device memroy 
	// to shared memory; each thread loads 
	// one element of each matrix

#pragma unroll

	for (int i = 0; i < wA; i++){
		As[i] = a[bx * wA + i];

	}

#pragma unroll

	for (int i = 0; i < wA;i++){
		    Bs[ty * wA + i] = b[by * BLOCK_SIZE * wB + ty * wB + i];
	}
	
	// Synchronize to make sure the matrices are loaded

	__syncthreads();

#pragma unroll
	for (int i = 0; i < wA; i++){
		float dif_ = As[i] - Bs[ty * wA + i];
			Csub += dif_ * dif_;		
	}

	// Synchronized to make sure that the preceeding 
	// computation is done 

	__syncthreads();
	// Write the block sub- matrix to device memory;
	// eahc thread writes one element

	int c_line = bx ;
	int c_col =  by * BLOCK_SIZE + ty;
	c[c_line * hB + c_col] = Csub;	
}

int distanceCompuation(int block_size, dim3 &dimsA, dim3 &dimsB, float *matrix_A, float *matrix_B, float *matrix_C, float *matrix_D);

int initCuda(){
	int devID = 0;

	hipSetDevice(devID);

	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	if (error != hipSuccess){
		printf("hipGetDevice returned error %s (code %d), line (%d)\n", hipGetErrorString(error), error, __LINE__);
	}

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited){
		fprintf(stderr, "Error: device is runing in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess){
		printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);

	}
	else{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}

	// Use a larger block size for Fermi and above
	int block_size = deviceProp.major < 2 ? 16 : 32;
	return block_size;
}

int main(int argc, char *argv[])
{


	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };
	int DIMENSIONS = 2;

	int num_samplingPoints; // �����������
	int width_originalPoints ; // ԭʼ������ĺ����������
	int height_originalPoints; // ԭʼ������������������
	int num_originalPoints; //
	float *m_samplingPoints; // �������λ�ã�
	float *m_originalPoints; // ԭʼ�ܶȺ�������ɢ�㣻
	float *m_samplingPointsDensity; // ��������ܶȣ�
	float *m_originalPointsDesntiy;

	num_samplingPoints = setNumSamplingPoint();
	width_originalPoints = setWidthOriginalPoint();
	height_originalPoints = setHeightOriginalPoint();
	num_originalPoints = width_originalPoints * height_originalPoints;

	m_samplingPoints = (float *)malloc(num_samplingPoints * DIMENSIONS * sizeof(float));
	m_originalPoints = (float *)malloc(width_originalPoints * height_originalPoints * DIMENSIONS * sizeof(float));
	m_samplingPointsDensity = (float *)malloc(num_samplingPoints * sizeof(float));
	m_originalPointsDesntiy = (float *)malloc(num_originalPoints * sizeof(float));

	setSamplingPoints(m_samplingPoints, num_samplingPoints, DIMENSIONS);
	setOriginalPoints(m_originalPoints, height_originalPoints, width_originalPoints, DIMENSIONS);
	setSamplingPointDensity(m_samplingPointsDensity, num_samplingPoints);
	setSamplingPointDensity(m_originalPointsDesntiy, num_originalPoints);	

	int block_size;

	// ��ʼ��CUDA
	block_size = initCuda();
	
	dim3 dims_sampling_points(num_samplingPoints, 2, 1);
	dim3 dims_original_points(width_originalPoints * height_originalPoints, 2, 1);
    // Add vectors in parallel.
	
	int _result = distanceCompuation(block_size, dims_sampling_points, dims_original_points, m_samplingPoints, m_originalPoints, m_samplingPointsDensity, m_originalPointsDesntiy);

	free(m_samplingPoints);
	free(m_originalPoints);
	free(m_originalPointsDesntiy);
	free(m_samplingPointsDensity);
	exit(_result);

	
}

int distanceCompuation(int block_size, dim3 &dimsA, dim3 &dimsB, float *matrix_A, float *matrix_B, float *matrix_C,float *matrix_D){
	// allocate host memory for original points and sampling points
	
	unsigned int size_A = dimsA.x * dimsA.y * dimsA.z;
	unsigned int mem_sizeA = sizeof(float) * size_A;
	float *h_A = matrix_A; // �����������

	unsigned int size_B = dimsB.x * dimsB.y * dimsB.z;
	unsigned int mem_sizeB = sizeof(float) * size_B;
	float  *h_B = matrix_B; // ������������꣬ԭʼͼ�������

	unsigned int size_samplingPoint = dimsA.x;
	unsigned int mem_sizeSamplingPoint = sizeof(float) * size_samplingPoint;
	float *h_samplingPointDensity = matrix_C;

	unsigned int size_originalPoint = dimsB.x;
	unsigned int mem_sizeOriginalPoint = sizeof(float) * size_originalPoint;
	float *h_originalPointDensity = matrix_D;

	unsigned int size_transportMatrix = dimsA.x * dimsB.x;
	unsigned int mem_sizeTransportMatrix = size_transportMatrix * sizeof(float);

	float *h_V = (float *)malloc(dimsB.x * sizeof(float));// ���㴫��ƻ�������v��
	for (int i = 0; i < dimsB.x; i++){
		*(h_V + i) = 1;
	}

	float *h_distanceMatrix;
	float *h_kasaiMatrix; // �������ĸ�˹����
	float *h_transportPlan;// = (float *)malloc(dimsA.x * dimsB.x * sizeof(float));

	// allocate device memory 
	float *d_A, *d_B, *d_distanceMatrix, *d_kasaiMatrix, *d_transportPlan, *d_U, *d_V; // device memory�еı���������d_U, d_V Ϊ�м����
	float *d_kasaiV, *d_kasaiU;
	float *d_samplingPointDensity, *d_originalPointDensity;
	float *d_diagUKasaiMatrix; /// ��ʱ����
	float *d_transportPlanDensity;///��ʱ����
	float *d_tempSamplPointCoordinate; // ��ʱ����
	hipblasHandle_t handle;
	hipblasStatus_t stat;
	hipError_t error;

	
	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS){
		printf("CUBLAS initialization failed\n");
		exit(EXIT_FAILURE);
	}
	
	error = hipMalloc((void**)&d_A, mem_sizeA);

	if (error != hipSuccess){
		printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString, error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_B, mem_sizeB);

	if (error != hipSuccess){
		printf("hipMalloc d_B returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_distanceMatrix, mem_sizeTransportMatrix);
	if (error != hipSuccess){
		printf("hipMalloc d_distanceMatrix returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_kasaiMatrix, mem_sizeTransportMatrix);
	if (error != hipSuccess){
		printf("hipMalloc d_kasaiMatrix returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_U, mem_sizeSamplingPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_U returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_V, mem_sizeOriginalPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_V returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}
	
	error = hipMalloc((void**)&d_kasaiV, mem_sizeSamplingPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_kasaiV returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_kasaiU, mem_sizeOriginalPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_kasaiU returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_samplingPointDensity, mem_sizeSamplingPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_samplingPointDensity returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_originalPointDensity, mem_sizeOriginalPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_originalPointDensity returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_diagUKasaiMatrix, mem_sizeTransportMatrix);
	if (error != hipSuccess){
		printf("hipMalloc d_diagUKasaiMatrix returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_transportPlan, mem_sizeTransportMatrix);
	if (error != hipSuccess){
		printf("hipMalloc d_transportPlan returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_transportPlanDensity, mem_sizeTransportMatrix);
	if (error != hipSuccess){
		printf("hipMalloc d_transportPlanDensity returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_tempSamplPointCoordinate, mem_sizeA);
	if (error != hipSuccess){
		printf("hipMalloc d_tempSamplPointCoordinate returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// copy host memory to device
	error = hipMemcpy(d_A, h_A, mem_sizeA, hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("hipMemcpy (d_A, h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_B, h_B, mem_sizeB, hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("hipMemcpy (d_B, h_B) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_V, h_V, mem_sizeOriginalPoint, hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("hipMemcpy (d_V, h_V) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_samplingPointDensity, h_samplingPointDensity, mem_sizeSamplingPoint, hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("hipMemcpy (d_samplingPointDensity, h_samplingPointDensity) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_originalPointDensity, h_originalPointDensity, mem_sizeOriginalPoint, hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("hipMemcpy (d_originalPointDensity, h_originalPointDensity) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

   // ����event
	hipEvent_t start;
	error = hipEventCreate(&start);

	if (error != hipSuccess){
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess){
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess){
		fprintf(stderr, " Failed to record start evern (error code %s)! \n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Setup execution parameters
	block_size = 4;
	dim3 threads(1, block_size, 1);
	dim3 grid(dimsA.x, dimsB.x / block_size, 1);
	
	// ����������
	if (block_size == 16){
		distancePointToPointCUDA<4> << <grid, threads >> >(d_distanceMatrix, d_A, d_B, dimsA.x, dimsA.y, dimsB.x, dimsB.y);
	}
	else{
		distancePointToPointCUDA<4> << <grid, threads >> >(d_distanceMatrix, d_A, d_B, dimsA.x, dimsA.y, dimsB.x, dimsB.y);
	}
	/// ����distancePointToPointCUDA�Ƿ���ȷ
	h_distanceMatrix = (float *)malloc(mem_sizeTransportMatrix);
	error = hipMemcpy(h_distanceMatrix, d_distanceMatrix, mem_sizeTransportMatrix, hipMemcpyDeviceToHost);
	if (error != hipSuccess){
		printf("hipMemcpy (h_distanceMatrix, d_distanceMatrix) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}
	// cpu ����������
	float *c_C = (float *)malloc(mem_sizeTransportMatrix);
	for (int i = 0; i < dimsA.x; i++){
		for (int j = 0; j < dimsB.x; j++){
			float diff_x;
			float diff_y;
			diff_x = *(h_A + i * dimsA.y) - *(h_B + j*dimsB.y);
			diff_y = *(h_A + i * dimsA.y + 1) - *(h_B + j*dimsB.y + 1);
			*(c_C + dimsB.x * i + j) = diff_x * diff_x + diff_y * diff_y;
		}
	}

	float diff_ = 0;
	for (int i = 0; i < dimsA.x; i++){
		for (int j = 0; j < dimsB.x; j++)
			diff_ += abs(h_distanceMatrix[i * dimsB.x + j] - c_C[i*dimsB.x + j]);

	}
	printf("The difference distance computation between results of CPU and GPU is %f.\n", diff_);
	free(c_C);

	// ����Kasai����
	threads.x = block_size;
	threads.y = 1;
	threads.z = 1;
	grid.x = size_transportMatrix / threads.x;
	grid.y = 1;
	grid.z = 1;

	if (block_size == 16){
		kaisaiMatrixComputation<4> <<<grid, threads>> >(d_kasaiMatrix, d_distanceMatrix);
	}
	else{
		kaisaiMatrixComputation<4> <<<grid, threads >> >(d_kasaiMatrix, d_distanceMatrix);
	}
	/// �����Ƿ���ȷ
	h_kasaiMatrix = (float *)malloc(mem_sizeTransportMatrix);
	error = hipMemcpy(h_kasaiMatrix, d_kasaiMatrix, mem_sizeTransportMatrix, hipMemcpyDeviceToHost);
	if (error != hipSuccess){
		printf("hipMemcpy (h_distanceMatrix, d_distanceMatrix) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}
	float *check_kasaiMatrix = (float *)malloc(mem_sizeTransportMatrix);
	printf("kasaiMatrix on GPU and CPU \n ");
	for (int i = 0; i < size_transportMatrix; i++){
		check_kasaiMatrix[i] = exp(-h_distanceMatrix[i]);
		printf("%f  %f \n", h_kasaiMatrix[i], check_kasaiMatrix[i]);
		
	}
	diff_ = 0;
	for (int i = 0; i < size_transportMatrix; i++){
		diff_ += abs(check_kasaiMatrix[i]- h_kasaiMatrix[i]);
	}
	printf("The difference between kasai Matrix of CPU and GPU is %f.\n", diff_);
	free(check_kasaiMatrix);

	// ���㴫��ƻ�����
	 float alpha = 1.0 ;
	 float beta = 0.0 ;	 

	unsigned int _iter = 1;
	for (int i = 0; i < _iter; i++){	
		
		stat = hipblasSgemv(handle, HIPBLAS_OP_T, size_originalPoint, size_samplingPoint, &alpha, d_kasaiMatrix, size_originalPoint, d_V, 1, &beta, d_kasaiV,1);			
		if (stat != HIPBLAS_STATUS_SUCCESS){
			printf("hipblasSdot failed\n");
			exit(EXIT_FAILURE);
		}
		
		threads.x = 2; //block_size;
		threads.y = 1;
		threads.z = 1;
		grid.x = size_samplingPoint / threads.x;
		grid.y = 1;
		grid.z = 1;
		elementWiseDIV<2> << <grid, threads >> >(d_U, d_samplingPointDensity, d_kasaiV);
		stat = hipblasSgemv(handle, HIPBLAS_OP_N, size_originalPoint, size_samplingPoint, &alpha, d_kasaiMatrix, size_originalPoint, d_U, 1, &beta, d_kasaiU, 1);
		if (stat != HIPBLAS_STATUS_SUCCESS){
			printf("hipblasSdot failed\n");
			exit(EXIT_FAILURE);
		}
		threads.x = block_size;
		threads.y = 1;
		threads.z = 1;
		grid.x = size_originalPoint / threads.x;
		grid.y = 1;
		grid.z = 1;
		elementWiseDIV<4> << <grid, threads >> >(d_V, d_originalPointDensity, d_kasaiU);
	}
	
	float *h_kasaiV = (float *)malloc(size_samplingPoint);
	error = hipMemcpy(h_kasaiV, d_kasaiV, mem_sizeSamplingPoint, hipMemcpyDeviceToHost);
	if (error != hipSuccess){
		printf("hipMemcpy (h_kasaiV, d_kasaiV) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}
	
	float *h_U = (float *)malloc(mem_sizeSamplingPoint);
	error = hipMemcpy(h_U, d_U, mem_sizeSamplingPoint, hipMemcpyDeviceToHost);
	if (error != hipSuccess){
		printf("hipMemcpy (h_U, d_U) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	float *h_kasaiU = (float *)malloc(mem_sizeOriginalPoint);
	error = hipMemcpy(h_kasaiU, d_kasaiU, mem_sizeOriginalPoint, hipMemcpyDeviceToHost);
	if (error != hipSuccess){
		printf("hipMemcpy (h_kasaiU, d_kasaiU) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(h_V, d_V, mem_sizeOriginalPoint, hipMemcpyDeviceToHost);
	if (error != hipSuccess){
		printf("hipMemcpy (h_V, d_V) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}
	

	// �����ȷ��
	float *check_kasaiV = (float *)malloc(mem_sizeSamplingPoint);
	printf("kasaiV vector: GPU  CPU\n");
	for (int i = 0; i < size_samplingPoint; i++){
		float temp_ = 0;
		for (int j = 0; j < size_originalPoint; j++){
			temp_ += h_kasaiMatrix[i*size_originalPoint + j] * h_V[j];
		}
		check_kasaiV[i] = temp_;
		printf("KasaiV vector: %f  %f\n", h_kasaiV[i], check_kasaiV[i]);
	}
	diff_ = 0;
	for (int i = 0; i < size_samplingPoint; i++){
		diff_ += abs(h_kasaiV[i] - check_kasaiV[i]);
	}
	printf("The differenc of kasaiV vector: %f\n", diff_);
	free(check_kasaiV);

	float *check_U = (float *)malloc(mem_sizeSamplingPoint);
	printf("temp vector U: GPU    CPU\n");
	for (int i = 0; i < size_samplingPoint; i++){
		check_U[i] = h_samplingPointDensity[i] / h_kasaiV[i];
		printf("%f  %f\n", h_U[i], check_U[i]);
	}

	diff_ = 0;
	for (int i = 0; i < size_samplingPoint; i++){
		diff_ += abs(h_U[i] - check_U[i]);
	}
	printf("the difference of vector U: %f\n", diff_);

	free(check_U);


	float *check_kasaiU = (float *)malloc(mem_sizeOriginalPoint);
	printf("kasaiU vector: GPU  CPU\n");
	for (int i = 0; i < size_originalPoint; i++){
		float temp_ = 0;
		for (int j = 0; j < size_samplingPoint; j++){
			temp_ += h_kasaiMatrix[j*size_originalPoint + i] * h_U[j];
		}
		check_kasaiU[i] = temp_;
		printf("KasaiU vector: %f  %f\n", h_kasaiU[i], check_kasaiU[i]);
	}
	diff_ = 0;
	for (int i = 0; i < size_originalPoint; i++){
		diff_ += abs(h_kasaiU[i] - check_kasaiU[i]);
	}
	printf("The differenc of kasaiU vector: %f\n", diff_);
	free(check_kasaiU);


	float *check_V = (float *)malloc(mem_sizeOriginalPoint);
	printf("temp vector V: GPU    CPU\n");
	for (int i = 0; i < size_originalPoint; i++){
		check_V[i] = h_originalPointDensity[i] / h_kasaiU[i];
		printf("%f  %f\n", h_V[i], check_V[i]);
	}

	diff_ = 0;
	for (int i = 0; i < size_originalPoint; i++){
		diff_ += abs(h_V[i] - check_V[i]);
	}
	printf("the difference of vector V: %f\n", diff_);

	free(check_V);

	///���㴫��ƻ�����
	// ����hipblasSdgmm�����Ծ���û��op���������������൱��ת�õļ��㣬 A= BCD   AT = DT CT BT (T��ʾת��)�ر�С��
	// �Դ��м������ d_transportMatrix ���󣬸պ���һ�� size_samplingPoint * size_originalPoint �Ҹպ��ǰ��������õľ���
	hipblasSdgmm(handle, HIPBLAS_SIDE_LEFT, size_originalPoint, size_samplingPoint, d_kasaiMatrix, size_originalPoint, d_V, 1, d_diagUKasaiMatrix, size_originalPoint);
	hipblasSdgmm(handle, HIPBLAS_SIDE_RIGHT, size_originalPoint, size_samplingPoint, d_diagUKasaiMatrix, size_originalPoint, d_U, 1, d_transportPlan, size_originalPoint);

	// �˶���ȷ��

	h_transportPlan = (float *)malloc(mem_sizeTransportMatrix);
	
	error = hipMemcpy(h_transportPlan, d_transportPlan, mem_sizeTransportMatrix, hipMemcpyDeviceToHost);
	if (error != hipSuccess){
		printf("hipMemcpy (h_transportPlan, d_transportPlan) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	float *check_transportPlan = (float *)malloc(mem_sizeTransportMatrix);
	float *ch_transportPlan = (float *)malloc(mem_sizeTransportMatrix);
	for (int i = 0; i < size_samplingPoint; i++){
		for (int j = 0; j < size_originalPoint; j++){
			ch_transportPlan[i*size_originalPoint + j] = h_U[i] * h_kasaiMatrix[i*size_originalPoint + j];
		}
	}
	for (int i = 0; i < size_samplingPoint; i++){
		for (int j = 0; j < size_originalPoint; j++){
			check_transportPlan[i*size_originalPoint + j] = ch_transportPlan[i*size_originalPoint + j] * h_V[j];
		}
	}

	printf("Transport plan matrix: GPU\n");
	for (int i = 0; i < size_samplingPoint; i++){
		for (int j = 0; j < size_originalPoint; j++){
			printf("  %f  ", h_transportPlan[i*size_originalPoint + j]);
		}
		printf("\n");
	}


	printf("Transport plan matrix: CPU\n");
	for (int i = 0; i < size_samplingPoint; i++){
		for (int j = 0; j < size_originalPoint; j++){
			printf("  %f  ", check_transportPlan[i*size_originalPoint + j]);
		}
		printf("\n");
	}

	free(check_transportPlan);
	free(ch_transportPlan);

	/// ��������ֵ����
	hipblasSdgmm(handle, HIPBLAS_SIDE_RIGHT, size_originalPoint, size_samplingPoint, d_transportPlan, size_originalPoint, d_samplingPointDensity, 1, d_transportPlanDensity, size_originalPoint);
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dimsA.y, size_samplingPoint, size_originalPoint, &alpha, d_B, dimsA.y, d_transportPlanDensity, size_originalPoint, &beta, d_tempSamplPointCoordinate, dimsA.y);

	float *h_transportPlanDensity = (float *)malloc(mem_sizeTransportMatrix);
	error = hipMemcpy(h_transportPlanDensity, d_transportPlanDensity, mem_sizeTransportMatrix, hipMemcpyDeviceToHost);
	if (error != hipSuccess){
		printf("hipMemcpy (h_transportPlanDensity, d_transportPlanDensity) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}
	
	float *h_tempSamplPointCoordinate = (float *)malloc(mem_sizeA);
	error = hipMemcpy(h_tempSamplPointCoordinate, d_tempSamplPointCoordinate, mem_sizeA, hipMemcpyDeviceToHost);
	if (error != hipSuccess){
		printf("hipMemcpy (h_tempSamplPointCoordinate, d_tempSamplPointCoordinate) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// �˶���ȷ��
	float *h_transportPlanDensityT = (float *)malloc(mem_sizeTransportMatrix);
	for (int i = 0; i < size_originalPoint; i++){
		for (int j = 0; j < size_samplingPoint; j++){
			h_transportPlanDensityT[j*size_originalPoint + i] = h_transportPlanDensity[i*size_samplingPoint + j];
		}
	}


	float *check_transportPlanDensity = (float *)malloc(mem_sizeTransportMatrix);
	float *check_tempSamplPointCoordinate = (float *)malloc(mem_sizeA);
	float *h_transportPlanT = (float *)malloc(mem_sizeTransportMatrix);
	for (int i = 0; i < size_samplingPoint; i++){
		for (int j = 0; j < size_originalPoint; j++){
			h_transportPlanT[j*size_samplingPoint +i] = h_transportPlan[i*size_originalPoint + j];
		}
	}
	for (int i = 0; i < size_originalPoint; i++){
		for (int j = 0; j < size_samplingPoint; j++){
			check_transportPlanDensity[i*size_samplingPoint + j] = h_transportPlanT[i*size_samplingPoint + j] * h_samplingPointDensity[j];
		}
	}

	printf("Tempt matrix h_transportPlanDensity: GPU   CPU\n");
	for (int i = 0; i < size_samplingPoint; i++){
		for (int j = 0; j < size_originalPoint; j++){
			printf("  %f   ", h_transportPlanDensityT[i*size_originalPoint + j]);
		}
		printf("\n");
	}

	for (int i = 0; i < size_samplingPoint; i++){
		for (int j = 0; j < size_originalPoint; j++){
			printf("  %f   ", check_transportPlanDensity[i*size_originalPoint + j]);
		}
		printf("\n");
	}

	// CUBLAS handle
	
	stat = hipblasDestroy(handle);
	if (stat != HIPBLAS_STATUS_SUCCESS){
		printf("hipblasDestroy failed\n");
		exit(EXIT_FAILURE);
	}	

	// Record the stop event
	error = hipEventRecord(stop, NULL);
	if (error != hipSuccess){
		fprintf(stderr, " Failed to record stop event ( error code %s)! \n", hipGetErrorString(error));
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);

	if (error != hipSuccess){
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)! \n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	
	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);

	if (error != hipSuccess){
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	
	for (int i = 0; i < dimsA.x; i++){
		for (int j = 0; j < dimsA.y; j++){	
			printf("%f  ", h_A[j*dimsA.x + i]);
		}
		printf("\n");
	}

	for (int i = 0; i < dimsA.x * dimsA.y; i++){
		
			printf("%f  ", h_A[ i]);
		}
		printf("\n");

	
	for (int i = 0; i < dimsB.x; i++){
		for (int j = 0; j < dimsB.y; j++){		
			printf("%f  ", h_B[ i * dimsB.y + j]);
		}
		printf("\n");
	}

	for (int i = 0; i < dimsA.x; i++){
		for (int j = 0; j < dimsB.x; j++){
	//		printf("%f  ", h_C[i*dimsB.x + j]);
		}
		printf("\n");
	}

	for (int i = 0; i < dimsB.x; i++){
		for (int j = 0; j < dimsB.y; j++){
//			printf("%f  ", h_D[i * dimsB.y + j]);
		}
		printf("\n");
	}
	
	// Check  the result
	/*
	float *c_C = (float *)malloc(mem_sizeC);
	for (int i = 0; i < dimsA.x; i++){
		for (int j = 0; j < dimsB.x; j++){
			float diff_x;
			float diff_y;
			diff_x = *(h_A + i * dimsA.y) - *(h_B + j*dimsB.y );
			diff_y = *(h_A + i * dimsA.y + 1) - *(h_B + j*dimsB.y + 1);
			*(c_C + dimsB.x * i + j) = diff_x * diff_x + diff_y * diff_y;
		}
	}

	float diff_ = 0;
	for (int i = 0; i < dimsA.x; i++){
		for (int j = 0; j < dimsB.x; j++)
			diff_ += ( h_C[i * dimsB.x + j] - c_C[i*dimsB.x + j]) * (h_C[i * dimsB.x + j] - c_C[i*dimsB.x + j] );

	}
	*/

	//printf("The difference between results of CPU and GPU is %f.\n", diff_);

	//printf("The dot product of h_B is %f\n", result_);

	//Clean up memory
	//free(h_A);
	//free(h_B);
	//free(h_C);
	//free(h_D);
	free(h_transportPlan);
	hipFree(d_A);
	hipFree(d_B);
	//hipFree(d_C);

	printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits

	hipDeviceReset();

	return EXIT_SUCCESS;

}


// set variables for distance compuation between points
void setVariableForDistanceComputation(){




	return;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
	int *dev_d = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_d, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);



	addKernel << <1, size >> >(dev_d, dev_c, dev_b);


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_d, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
	
	hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

/*
hipError_t multiMatriWithCuda(float *c, float *a, float *b, int widthA, int heightA, int widthB, int heightB)
{
	float2 *dev_a = 0;
	float2 *dev_b = 0;
	float2 *dev_c = 0;
	
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, heightA * heightB * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, heightA * widthA * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, heightB * widthB * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}



	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, widthA * heightA * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, widthB * heightB * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_d, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
*/